/*
 * Academic License - for use in teaching, academic research, and meeting
 * course requirements at degree granting institutions only.  Not for
 * government, commercial, or other organizational use.
 *
 * fog_rectification_terminate.cu
 *
 * Code generation for function 'fog_rectification_terminate'
 *
 */

/* Include files */
#include "rt_nonfinite.h"
#include "fog_rectification.h"
#include "fog_rectification_terminate.h"
#include "_coder_fog_rectification_mex.h"
#include "fog_rectification_data.h"

/* Function Definitions */
void fog_rectification_atexit(void)
{
  mexFunctionCreateRootTLS();
  emlrtEnterRtStackR2012b(emlrtRootTLSGlobal);
  emlrtLeaveRtStackR2012b(emlrtRootTLSGlobal);
  emlrtDestroyRootTLS(&emlrtRootTLSGlobal);
}

void fog_rectification_terminate(void)
{
  emlrtLeaveRtStackR2012b(emlrtRootTLSGlobal);
  emlrtDestroyRootTLS(&emlrtRootTLSGlobal);
}

/* End of code generation (fog_rectification_terminate.cu) */
