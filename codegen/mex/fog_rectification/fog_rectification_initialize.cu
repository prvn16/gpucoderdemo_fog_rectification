/*
 * Academic License - for use in teaching, academic research, and meeting
 * course requirements at degree granting institutions only.  Not for
 * government, commercial, or other organizational use.
 *
 * fog_rectification_initialize.cu
 *
 * Code generation for function 'fog_rectification_initialize'
 *
 */

/* Include files */
#include "rt_nonfinite.h"
#include "fog_rectification.h"
#include "fog_rectification_initialize.h"
#include "_coder_fog_rectification_mex.h"
#include "fog_rectification_data.h"

/* Function Definitions */
void fog_rectification_initialize(void)
{
  mexFunctionCreateRootTLS();
  emlrtClearAllocCountR2012b(emlrtRootTLSGlobal, false, 0U, 0);
  emlrtEnterRtStackR2012b(emlrtRootTLSGlobal);
  emlrtLicenseCheckR2012b(emlrtRootTLSGlobal, "Image_Toolbox", 2);
  emlrtFirstTimeR2012b(emlrtRootTLSGlobal);
}

/* End of code generation (fog_rectification_initialize.cu) */
