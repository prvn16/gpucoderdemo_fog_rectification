/*
 * Academic License - for use in teaching, academic research, and meeting
 * course requirements at degree granting institutions only.  Not for
 * government, commercial, or other organizational use.
 *
 * _coder_fog_rectification_api.cu
 *
 * Code generation for function '_coder_fog_rectification_api'
 *
 */

/* Include files */
#include "rt_nonfinite.h"
#include "fog_rectification.h"
#include "_coder_fog_rectification_api.h"
#include "fog_rectification_data.h"

/* Variable Definitions */
static const int32_T iv0[3] = { 480, 640, 3 };

/* Function Declarations */
static uint8_T (*b_emlrt_marshallIn(const mxArray *u, const emlrtMsgIdentifier
  *parentId))[921600];
static uint8_T (*c_emlrt_marshallIn(const mxArray *src, const emlrtMsgIdentifier
  *msgId))[921600];
static uint8_T (*emlrt_marshallIn(const mxArray *input, const char_T *identifier))
  [921600];
static const mxArray *emlrt_marshallOut(const uint8_T u[921600]);

/* Function Definitions */
static uint8_T (*b_emlrt_marshallIn(const mxArray *u, const emlrtMsgIdentifier
  *parentId))[921600]
{
  uint8_T (*y)[921600];
  y = c_emlrt_marshallIn(emlrtAlias(u), parentId);
  emlrtDestroyArray(&u);
  return y;
}
  static uint8_T (*c_emlrt_marshallIn(const mxArray *src, const
  emlrtMsgIdentifier *msgId))[921600]
{
  uint8_T (*ret)[921600];
  emlrtCheckBuiltInR2012b(emlrtRootTLSGlobal, msgId, src, "uint8", false, 3U,
    iv0);
  ret = (uint8_T (*)[921600])emlrtMxGetData(src);
  emlrtDestroyArray(&src);
  return ret;
}

static uint8_T (*emlrt_marshallIn(const mxArray *input, const char_T *identifier))
  [921600]
{
  uint8_T (*y)[921600];
  emlrtMsgIdentifier thisId;
  thisId.fIdentifier = (const char *)identifier;
  thisId.fParent = NULL;
  thisId.bParentIsCell = false;
  y = b_emlrt_marshallIn(emlrtAlias(input), &thisId);
  emlrtDestroyArray(&input);
  return y;
}
  static const mxArray *emlrt_marshallOut(const uint8_T u[921600])
{
  const mxArray *y;
  const mxArray *m0;
  static const int32_T iv2[3] = { 0, 0, 0 };

  y = NULL;
  m0 = emlrtCreateNumericArray(3, iv2, mxUINT8_CLASS, mxREAL);
  emlrtMxSetData((mxArray *)m0, (void *)&u[0]);
  emlrtSetDimensions((mxArray *)m0, iv0, 3);
  emlrtAssign(&y, m0);
  return y;
}

void fog_rectification_api(fog_rectificationStackData *SD, const mxArray * const
  prhs[1], const mxArray *plhs[1])
{
  uint8_T (*out)[921600];
  uint8_T (*input)[921600];
  out = (uint8_T (*)[921600])mxMalloc(sizeof(uint8_T [921600]));

  /* Marshall function inputs */
  input = emlrt_marshallIn(emlrtAlias(prhs[0]), "input");

  /* Invoke the target function */
  fog_rectification(SD, *input, *out);

  /* Marshall function outputs */
  plhs[0] = emlrt_marshallOut(*out);
}

/* End of code generation (_coder_fog_rectification_api.cu) */
