/*
 * Academic License - for use in teaching, academic research, and meeting
 * course requirements at degree granting institutions only.  Not for
 * government, commercial, or other organizational use.
 *
 * _coder_fog_rectification_mex.cu
 *
 * Code generation for function '_coder_fog_rectification_mex'
 *
 */

/* Include files */
#include "fog_rectification.h"
#include "_coder_fog_rectification_mex.h"
#include "fog_rectification_terminate.h"
#include "_coder_fog_rectification_api.h"
#include "fog_rectification_initialize.h"
#include "fog_rectification_data.h"

/* Function Declarations */
static void fog_rectification_mexFunction(fog_rectificationStackData *SD,
  int32_T nlhs, mxArray *plhs[1], int32_T nrhs, const mxArray *prhs[1]);

/* Function Definitions */
static void fog_rectification_mexFunction(fog_rectificationStackData *SD,
  int32_T nlhs, mxArray *plhs[1], int32_T nrhs, const mxArray *prhs[1])
{
  int32_T n;
  const mxArray *inputs[1];
  const mxArray *outputs[1];
  int32_T b_nlhs;

  /* Check for proper number of arguments. */
  if (nrhs != 1) {
    emlrtErrMsgIdAndTxt(emlrtRootTLSGlobal, "EMLRT:runTime:WrongNumberOfInputs",
                        5, 12, 1, 4, 17, "fog_rectification");
  }

  if (nlhs > 1) {
    emlrtErrMsgIdAndTxt(emlrtRootTLSGlobal,
                        "EMLRT:runTime:TooManyOutputArguments", 3, 4, 17,
                        "fog_rectification");
  }

  /* Temporary copy for mex inputs. */
  for (n = 0; n < nrhs; n++) {
    inputs[n] = prhs[n];
  }

  /* Call the function. */
  fog_rectification_api(SD, inputs, outputs);

  /* Copy over outputs to the caller. */
  if (nlhs < 1) {
    b_nlhs = 1;
  } else {
    b_nlhs = nlhs;
  }

  emlrtReturnArrays(b_nlhs, plhs, outputs);

  /* Module termination. */
  fog_rectification_terminate();
}

void mexFunction(int32_T nlhs, mxArray *plhs[], int32_T nrhs, const mxArray
                 *prhs[])
{
  fog_rectificationStackData *c_fog_rectificationStackDataGlo = NULL;
  c_fog_rectificationStackDataGlo = (fog_rectificationStackData *)emlrtMxCalloc
    (1, 1U * sizeof(fog_rectificationStackData));
  mexAtExit(fog_rectification_atexit);

  /* Initialize the memory manager. */
  /* Module initialization. */
  fog_rectification_initialize();

  /* Dispatch the entry-point. */
  fog_rectification_mexFunction(c_fog_rectificationStackDataGlo, nlhs, plhs,
    nrhs, prhs);
  emlrtMxFree(c_fog_rectificationStackDataGlo);
}

emlrtCTX mexFunctionCreateRootTLS(void)
{
  emlrtCreateRootTLS(&emlrtRootTLSGlobal, &emlrtContextGlobal, NULL, 1);
  return emlrtRootTLSGlobal;
}

/* End of code generation (_coder_fog_rectification_mex.cu) */
